#include "hip/hip_runtime.h"
#include "agent.h"
#include "engine.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include "math.h"
#include "IOUtils.h"
#include <stdint.h>
#include <iostream>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

#ifdef __JETBRAINS_IDE__
#define __host__
#define __device__
#define __global__
#endif

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

constexpr float ALFA = M_PI/2;
constexpr int RESOLUTION = 180;
constexpr int RESOLUTION_SHIFT = RESOLUTION + 1;
constexpr float COLLISION_RADIUS = 100;
constexpr int MAX_BOARDS = 10000;
constexpr float ALFA_EPS = ALFA/RESOLUTION;
constexpr int MULTIPLIER = 1000000000/(10*MAX_BOARDS); 

__device__ vo compute_simple_vo(const agent& A, const agent& B, int agent_radius){
    vo obs;
    vec2 pAB = B.pos() - A.pos();
    obs.apex = A.pos() + B.svect() + (pAB.normalized()*agent_radius);
    float theta = asin(2 * agent_radius / pAB.length());
    obs.left = pAB.normalized().rotate(theta);
    obs.right = pAB.normalized().rotate(-theta);
    return obs;
}

__global__ void find_path(agent *agents, int n_agents, float agent_radius, float max_speed) {
    int ix = blockDim.x * blockIdx.x + threadIdx.x;
    if(ix >= n_agents)
        return;

    agent &james = agents[ix];
    james.set_vector((james.dest() - james.pos()).normalized());
    if(james.finished(agent_radius) or james.isdead())
        james.set_speed(0);
    else{
        james.set_speed(max_speed);
    }
}

__global__ void set_vo(agent *agents, vo *obstacles, int n_agents, int agent_radius) {
    int ix = blockDim.x * blockIdx.x + threadIdx.x;
    int i1 = ix / n_agents;
    int i2 = ix % n_agents;
    if (ix >= n_agents * n_agents || i1 == i2) 
        return;

    agent &A = agents[i1];
    agent &B = agents[i2];
    if(distance(A.pos(), B.pos()) < COLLISION_RADIUS){
        obstacles[ix] = compute_simple_vo(agents[i1], agents[i2], agent_radius);
        if(obstacles[ix].contains(A.pos()))
            obstacles[ix].set_invalid();
    }
    else
        obstacles[ix].set_invalid();
}

__global__ void clear_best_distances(int *best_distances, int rays_number){
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if(i >= rays_number)
        return;

    best_distances[i] = INT32_MAX;
}

__global__ void get_worst_intersects(agent *agents, vo *obstacles, int *best_distances, 
    unsigned long long *best_intersects, int n_agents, float max_speed){

    int ix = blockDim.x * blockIdx.x + threadIdx.x;
    int i1 = ix / n_agents;
    int i2 = ix % n_agents;

    if (ix >= n_agents * n_agents || i1 == i2 || obstacles[ix].invalid()) 
        return;

    vo &obs = obstacles[ix];
    agent &A = agents[i1];

    ray rays[2];
    rays[0] = obs.left_ray(); 
    rays[1] = obs.right_ray();
    vec2 left_angle = A.vect().rotate(ALFA/2);

    float distA = A.dist(); 
    vec2 p[2]; // points of intersection v_ray with angle
    float d[2]; // distance from p[i] to A.pos() - we need to get closest point
    

    for (int i = 0; i <= RESOLUTION; ++i){
        ray v_ray(A.pos(), left_angle.rotate(-i*ALFA_EPS));
        for (int j = 0; j < 2; ++j){
            p[j] = intersect_rays(rays[j], v_ray);
            if(p[j].invalid())
                p[j] = v_ray.pos + (v_ray.dir * max_speed);
            d[j] = min(max_speed, distance(p[j], A.pos()));
            p[j] = v_ray.pos + (v_ray.dir * d[j]);
        }

        if(d[1] < d[0]){
            p[0] = p[1];
            d[0] = d[1];
        }

        unsigned long long point;
        float* ptr = reinterpret_cast<float*>(&point);
        *ptr = p[0].x();
        *(ptr + 1) = p[0].y();
        int point_distance = d[0]*MULTIPLIER; // multiply to give approximation
        int old = atomicMin(&best_distances[RESOLUTION_SHIFT*i1 + i], point_distance);
        if(point_distance < old) // some minor 'swaps' may occur
            atomicExch(&best_intersects[RESOLUTION_SHIFT*i1 + i], point);

    }
    
}


__global__ void apply_best_velocities(agent *agents, int *best_distances, unsigned long long *intersects, int n_agents, float max_speed){
    int ai = blockDim.x * blockIdx.x + threadIdx.x;
    if (ai >= n_agents)
        return;

    agent &A = agents[ai];
    int best_dist = INT32_MAX;
    vec2 best_p, p;
    for (int i = 0; i <= RESOLUTION; ++i){
        if(best_distances[RESOLUTION_SHIFT*ai + i] == INT32_MAX){ // ray is free
           vec2 v = A.vect().rotate(ALFA/2).rotate(-i*ALFA_EPS);
           p = A.pos() + (v*max_speed);
        }
        else{
            float* ptr = reinterpret_cast<float*>(&intersects[RESOLUTION_SHIFT*ai + i]);
            p.set(*ptr, *(ptr+1));
        }
        float dist = distance(p, A.dest());
        if(dist < best_dist){
            best_dist = dist;
            best_p = p;
        }
    }

    best_p = best_p - A.pos();
    A.set_speed(best_p.length());
    A.set_vector(best_p.normalized());
}

__global__ void move(agent *agents, int n_agents, int move_divider) {
    int ix = blockDim.x * blockIdx.x + threadIdx.x;
    if (ix >= n_agents)
        return;

    agents[ix].move(move_divider);
}

void run(int n_agents, int n_generations, float agent_radius, float max_speed, int board_x, int board_y, int move_divider, agent* agents) {
    if(board_x > MAX_BOARDS || board_y > MAX_BOARDS)
        std::cout << "Exceeded MAX_BOARDS size. Bugs may occur\n";

    openFiles();
    putMetadataToFile(n_agents, n_generations, agent_radius, board_x, board_y);
    writeAgenstStartPosition(agents, n_agents);

    int rays_number = (n_agents * n_agents * RESOLUTION_SHIFT);
    int pairs_number = n_agents * n_agents;
    agent *d_agents;
    vo *d_obstacles;
    int *d_best_distances;
    unsigned long long *d_best_intersects;
    gpuErrchk(hipMalloc(&d_agents, n_agents * sizeof(agent)));
    gpuErrchk(hipMalloc(&d_obstacles, n_agents * n_agents * sizeof(vo)));
    gpuErrchk(hipMalloc(&d_best_distances, rays_number * sizeof(int)));
    gpuErrchk(hipMalloc(&d_best_intersects, rays_number * sizeof(long long)));

    gpuErrchk(hipMemcpy(d_agents, agents, n_agents * sizeof(agent), hipMemcpyHostToDevice));

    int block_size = 1024;
    int grid_size_agents = n_agents / block_size + 1;
    int grid_size_pairs = pairs_number / block_size + 1;
    int grid_size_rays = rays_number / block_size + 1;

    for (int i = 0; i < n_generations; ++i) {
        clear_best_distances<<<grid_size_rays, block_size>>>(d_best_distances, rays_number);
        gpuErrchk(hipDeviceSynchronize());
        find_path<<<grid_size_agents, block_size>>>(d_agents, n_agents, agent_radius, max_speed);
        gpuErrchk(hipDeviceSynchronize());

        set_vo<<<grid_size_pairs, block_size>>>(d_agents, d_obstacles, n_agents, agent_radius);
        gpuErrchk(hipDeviceSynchronize());
        get_worst_intersects<<<grid_size_pairs, block_size>>>(d_agents, d_obstacles, d_best_distances, d_best_intersects, n_agents, max_speed);
        gpuErrchk(hipDeviceSynchronize());

        apply_best_velocities<<<grid_size_agents, block_size>>>(d_agents, d_best_distances, d_best_intersects, n_agents, max_speed);
        gpuErrchk(hipDeviceSynchronize());
        move<<<grid_size_pairs, block_size>>>(d_agents, n_agents, move_divider);

        gpuErrchk(hipMemcpy(agents, d_agents, n_agents * sizeof(agent), hipMemcpyDeviceToHost));
        writeAgentsPositions(agents, n_agents);
    }

    closeFiles();
    hipFree(d_agents);
    hipFree(d_obstacles);
}